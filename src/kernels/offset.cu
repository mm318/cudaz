#include "hip/hip_runtime.h"
extern "C" __global__ void offset(tuple *in, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    out[i] = in[i].y - in[i].x;
}
